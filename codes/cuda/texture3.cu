#include "hip/hip_runtime.h"
/*
the char will map to [-1,1]

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


*/
#include <iostream>
using namespace std;

#define DIM 16
texture<char,1,hipReadModeNormalizedFloat>  texIn;


__global__ void fun(char * in, float * out)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //we can access to texture memory like normal global memory
	//out[tid] = in[tid] * 2;
	out[tid] = tex1Dfetch(texIn,tid);
}

int main()
{
char * inGlobe;
float * outGlobe;
char tmp[DIM][DIM];
float tmp2[DIM][DIM];

hipMalloc((void**)&inGlobe,DIM * DIM  );
hipMalloc((void**)&outGlobe,DIM * DIM * sizeof (float));
//hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
hipBindTexture( NULL, texIn,inGlobe,                
                 DIM * DIM * sizeof (char) );
	
for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
	{
		tmp[i][j] = i * DIM + j;
		//cout << tmp[i][j] << "  " ;
	}
	//cout << endl;
}

hipMemcpy( inGlobe, tmp, DIM * DIM, hipMemcpyHostToDevice );    

fun<<<DIM,DIM>>>(inGlobe,outGlobe);
	
hipMemcpy( tmp2, outGlobe, DIM * DIM * sizeof (float), hipMemcpyDeviceToHost );

for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
		cout << tmp2[i][j] << "  " ;
	cout << endl;
}
 int k;
 cin >> k;
return 0;
}



/*
the unsigned char will map to [0,1]
*/

#include <iostream>
using namespace std;

#define DIM 16
texture<unsigned char,1,hipReadModeNormalizedFloat>  texIn;


__global__ void fun(unsigned char * in, float * out)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //we can access to texture memory like normal global memory
	//out[tid] = in[tid] * 2;
	out[tid] = tex1Dfetch(texIn,tid);
}

int main()
{
unsigned char * inGlobe;
float * outGlobe;
unsigned char tmp[DIM][DIM];
float tmp2[DIM][DIM];

hipMalloc((void**)&inGlobe,DIM * DIM  );
hipMalloc((void**)&outGlobe,DIM * DIM * sizeof (float));
//hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
hipBindTexture( NULL, texIn,inGlobe,                
                 DIM * DIM * sizeof (unsigned char) );
	
for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
	{
		tmp[i][j] = i * DIM + j;
		//cout << tmp[i][j] << "  " ;
	}
	//cout << endl;
}

hipMemcpy( inGlobe, tmp, DIM * DIM, hipMemcpyHostToDevice );    

fun<<<DIM,DIM>>>(inGlobe,outGlobe);
	
hipMemcpy( tmp2, outGlobe, DIM * DIM * sizeof (float), hipMemcpyDeviceToHost );

for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
		cout << tmp2[i][j] << "  " ;
	cout << endl;
}
 int k;
 cin >> k;
return 0;
}

