#include "hip/hip_runtime.h"
/*
one dimension float texture with integet index access.
it is OK.
*/
#include <iostream>
using namespace std;
    
#define DIM 16
    texture<float>  texIn;
    texture<float>  texOut;
    texture<float>  texConstSrc;
    
    __global__ void fun(float * in, float * out)
    {
    
        int tid = threadIdx.x + blockIdx.x * blockDim.x;
        //we can access to texture memory like normal global memory
        //out[tid] = in[tid] * 2;
        out[tid] = tex1Dfetch(texIn,tid) * 5;
    }
    
    int main()
    {
    float * inGlobe;
    float * outGlobe;
    float tmp[DIM][DIM];
    
    hipMalloc((void**)&inGlobe,DIM * DIM * 4 );
    hipMalloc((void**)&outGlobe,DIM * DIM * 4);
    //hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
    hipBindTexture( NULL, texIn,inGlobe,                
                     DIM * DIM * sizeof (float) );
        
    for (int i = 0; i < DIM; ++i)
    {
        for (int j = 0; j < DIM; ++j)
        {
            tmp[i][j] = i + j + 0.125;
            cout << tmp[i][j] << "  " ;
        }
        cout << endl;
    }
    
    hipMemcpy( inGlobe, tmp, DIM * DIM * 4, hipMemcpyHostToDevice );    
    
    fun<<<DIM,DIM>>>(inGlobe,outGlobe);
        
    hipMemcpy( tmp, outGlobe, DIM * DIM * 4, hipMemcpyDeviceToHost );
    
    for (int i = 0; i < DIM; ++i)
    {
        for (int j = 0; j < DIM; ++j)
            cout << tmp[i][j] << "  " ;
        cout << endl;
    }
     int k;
     cin >> k;
    return 0;
    }




/*
two dimension float texture with integet index access.
it is OK.
*/


#include <iostream>
using namespace std;

#define DIM 16
texture<float,2>  texIn;
texture<float,2>  texOut;
texture<float,2>  texConstSrc;

__global__ void fun(float * in, float * out)
{

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
    //we can access to texture memory like normal global memory
	//out[tid] = in[tid] * 2;
	out[tid] = tex2D(texIn,threadIdx.x,blockIdx.x) * 3;
}

int main()
{
float * inGlobe;
float * outGlobe;
float tmp[DIM][DIM];

hipMalloc((void**)&inGlobe,DIM * DIM * 4 );
hipMalloc((void**)&outGlobe,DIM * DIM * 4);
hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
hipBindTexture2D( NULL, texIn,inGlobe,
                 desc, DIM, DIM,
                 DIM * sizeof (float) );
	
for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
	{
		tmp[i][j] = i + j;
		cout << tmp[i][j] << "  " ;
	}
	cout << endl;
}

hipMemcpy( inGlobe, tmp, DIM * DIM * 4, hipMemcpyHostToDevice );    

fun<<<DIM,DIM>>>(inGlobe,outGlobe);
	
hipMemcpy( tmp, outGlobe, DIM * DIM * 4, hipMemcpyDeviceToHost );

for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
		cout << tmp[i][j] << "  " ;
	cout << endl;
}
 int k;
 cin >> k;
return 0;
}

