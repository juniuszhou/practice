/*
example to show how to use stream and async method to make the data
transfer and kernel function executed concurrently.
*/

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define DIM 16
__shared__ int share[256];

__global__ void fun(int * in, int * out)
{	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;	
	*out = *in + 1000*tid;
}

int main()
{
int * inGlobe;
int * outGlobe;
int tmp[DIM*DIM];
int tmp2[DIM*DIM];

hipStream_t    stream;
HANDLE_ERROR( hipStreamCreate( &stream ) );

HANDLE_ERROR(hipMalloc((void**)&inGlobe, DIM * DIM * sizeof (int)));
HANDLE_ERROR(hipMalloc((void**)&outGlobe, DIM * DIM * sizeof (int)));
	
for (int i = 0; i < DIM*DIM; ++i)
	tmp[i] = i;	

for (int i = 0; i < DIM*DIM; ++i)
{
hipMemcpyAsync(inGlobe+i,tmp+i,sizeof(int),hipMemcpyHostToDevice,stream ) ;
//0 is device and stream is exec stream.
fun<<<1,1,0,stream>>>(inGlobe+i,outGlobe+i);
hipMemcpyAsync(tmp2+i,outGlobe+i,sizeof(int),hipMemcpyDeviceToHost,stream);
}

 HANDLE_ERROR( hipStreamSynchronize( stream ) );

for (int i = 0; i < DIM * DIM; ++i)	
		cout << tmp2[i] << "  " ;
	

 int k;
 cin >> k;
return 0;
}

