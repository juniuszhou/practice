
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err)
             << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


int main( void ) {
    hipDeviceProp_t  prop;
    int dev;

    HANDLE_ERROR( hipGetDevice( &dev ) );
    printf( "ID of current CUDA device:  %d\n", dev );

    memset( &prop, 0, sizeof( hipDeviceProp_t ) );
    prop.major = 1;
    prop.minor = 3;
    HANDLE_ERROR( hipChooseDevice( &dev, &prop ) );
    printf( "ID of CUDA device closest to revision 1.3:  %d\n", dev );

    HANDLE_ERROR( hipSetDevice( dev ) );
}


