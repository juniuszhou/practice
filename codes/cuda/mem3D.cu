#include "hip/hip_runtime.h"
/*
this example to show malloc 2D interface used to get a memory
optimized add padding then make sure alignment is better for fetch
*/

#include <common_functions.h>
#include <iostream>
using namespace std;

__global__ void MyKernel(hipPitchedPtr devPitchedPtr, int width, int height, int depth) 
{ 
    char* devPtr = devPitchedPtr.ptr; 
    size_t pitch = devPitchedPtr.pitch;
    size_t slicePitch = pitch * height; 

    for (int z = 0; z < depth; ++z) 
    { 
        char* slice = devPtr + z * slicePitch; 
        for (int y = 0; y < height; ++y) 
        { 
            float* row = (float*)(slice + y * pitch); 
            for (int x = 0; x < width; ++x) 
            { 
                float element = row[x]; 
            } 
        }
    } 
}


int main(int argc, char **argv)
{
   int width = 64, height = 64, depth = 64;

   //assign a 3D access mode
    //how about if we set depth as 1
   hipExtent extent = make_hipExtent(width * sizeof(float), height, depth);
   
   hipPitchedPtr devPitchedPtr;
   //get a block 3D memory
   hipMalloc3D(&devPitchedPtr, extent);
   MyKernel<<<100, 512>>>(devPitchedPtr, width, height, depth);
   cin >> width;
   return 0;
}

