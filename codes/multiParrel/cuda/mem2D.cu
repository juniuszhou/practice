/*
this example to show malloc 2D interface used to get a memory
optimized add padding then make sure alignment is better for fetch
*/


#include <hip/hip_runtime.h>
#include <common_functions.h>
#include <iostream>
using namespace std;

__global__ void MyKernel(char* devPtr, size_t pitch, int width, int height)
{ 
	for (int r = 0; r < height; ++r)
	{ 
		float* row = (float*)((char*)devPtr + r * pitch);
		for (int c = 0; c < width; ++c) 
		{
			float element = row[c]; 
		} 
	} 
}

int main(int argc, char **argv)
{
   int width = 64, height = 64; 
   char* devPtr; size_t pitch; 
   hipMallocPitch(&devPtr, &pitch, width * sizeof(char), height);
   cout << sizeof(float) << endl;
   cout << pitch << endl;
   MyKernel<<<100, 512>>>(devPtr, pitch, width, height);
   cin >> width;
   return 0;
}

