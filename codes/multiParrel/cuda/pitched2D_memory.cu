#include "hip/hip_runtime.h"
/*
this example is to show how to use pitched memory to complete
a task, that move the item in a matrix to other position.
move right 5, and move down 7.
*/
#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define checkCudaErrors( err ) (HandleError( err, __FILE__, __LINE__ ))

#define NUM_REPS 100  // number of repetitions performed  
#define TILE_DIM 16   // tile/block size

texture<float, 2, hipReadModeElementType> texRefPL;

__global__ void shiftPitchLinear(float* odata, int pitch, int width, int height, 
				 int shiftX, int shiftY)
{
  int xid = blockIdx.x * blockDim.x + threadIdx.x;
  int yid = blockIdx.y * blockDim.y + threadIdx.y;
  
  odata[yid*pitch+xid] = tex2D(texRefPL, 
			       (xid + shiftX)/(float)width, 
			       (yid + shiftY)/(float)height);
}

int
main( int argc, char** argv) 
{
  // set array size
  const int nx = 2045;  
  const int ny = 2045;

  // shifts applied to x and y data
  const int x_shift = 5;
  const int y_shift = 7;

  dim3 grid(nx/TILE_DIM, ny/TILE_DIM), threads(TILE_DIM, TILE_DIM);  
  int devID = 0;

  // CUDA events for timing
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  float *h_idata = (float*) malloc(sizeof(float)*nx*ny);
  float *h_odata = (float*) malloc(sizeof(float)*nx*ny);
  float *gold = (float*) malloc(sizeof(float)*nx*ny);

  for(int i = 0; i < nx*ny; ++i) h_idata[i] = (float) i;    

  float *d_idataPL;
  //pitch size in device
  size_t d_pitchBytes;

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<float>();

  checkCudaErrors(hipMallocPitch((void**) &d_idataPL, &d_pitchBytes, nx*sizeof(float), ny));  
  cout << d_pitchBytes << endl;

  float *d_odata;
  checkCudaErrors(hipMallocPitch( (void**) &d_odata, &d_pitchBytes, nx*sizeof(float), ny));
  cout << d_pitchBytes << endl;
  
  //here h_pitchBytes means pitch in host memory. there is not padding.
  size_t h_pitchBytes = nx*sizeof(float);

  checkCudaErrors(hipMemcpy2D(d_idataPL, d_pitchBytes, h_idata, h_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyHostToDevice)); 

  texRefPL.normalized = 1;
  texRefPL.filterMode = hipFilterModePoint;
  texRefPL.addressMode[0] = hipAddressModeWrap;
  texRefPL.addressMode[1] = hipAddressModeWrap;
  
  checkCudaErrors(hipBindTexture2D(0, &texRefPL, d_idataPL, &channelDesc, nx, ny, d_pitchBytes));  
  
  checkCudaErrors(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  checkCudaErrors(hipEventRecord(start, 0));
  
  shiftPitchLinear<<<grid, threads>>>(d_odata, (int)(d_pitchBytes/sizeof(float)), nx, ny, 
					x_shift, y_shift);
 
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
  float timePL;
  checkCudaErrors(hipEventElapsedTime(&timePL, start, stop));  

  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
			     nx*sizeof(float), ny, hipMemcpyDeviceToHost));
  

  checkCudaErrors(hipMemset2D(d_odata, d_pitchBytes, 0, nx*sizeof(float), ny));
  checkCudaErrors(hipEventRecord(start, 0));
 
  
  
  checkCudaErrors(hipEventRecord(stop, 0));
  checkCudaErrors(hipEventSynchronize(stop));
 
  checkCudaErrors(hipMemcpy2D(h_odata, h_pitchBytes, d_odata, d_pitchBytes, 
		     nx*sizeof(float), ny, hipMemcpyDeviceToHost));
  
    
  float bandwidthPL = 2.f*1000.f*nx*ny*sizeof(float)/(1.e+9f)/(timePL/NUM_REPS);
 
  printf("\nBandwidth (GB/s) for pitch linear: %.2e;\n", 
	 bandwidthPL);

  float fetchRatePL = nx*ny/1.e+6f/(timePL/(1000.0f*NUM_REPS));
 
  printf("\nTexture fetch rate (Mpix/s) for pitch linear: %.2e;\n", 
	 fetchRatePL);

  free(h_idata);
  free(h_odata);
  free(gold);

  checkCudaErrors(hipUnbindTexture(texRefPL));
  
  checkCudaErrors(hipFree(d_idataPL));
 
  checkCudaErrors(hipFree(d_odata));

  checkCudaErrors(hipEventDestroy(start));
  checkCudaErrors(hipEventDestroy(stop));
  
  hipDeviceReset();

  }

