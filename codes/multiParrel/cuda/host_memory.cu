#include "hip/hip_runtime.h"
/*
example to show how to use host memory in cuda func.
after allocate memory in host, cuda can operate it directly using pointer.
the result is transparent to host
*/

#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define DIM 16
__shared__ int share[256];

__global__ void fun(int * in, int * out)
{	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;	
	out[tid] = tid;
}

int main()
{
int * inGlobe;
int * outGlobe;
int tmp[DIM*DIM];
int * host_a;

HANDLE_ERROR(hipHostAlloc( (void**)&host_a,DIM * DIM * sizeof(int),hipHostMallocDefault ));

for (int i = 0; i < DIM*DIM; ++i)
{
	host_a[i] = i;	
} 

HANDLE_ERROR(hipMalloc((void**)&inGlobe, DIM * DIM * sizeof (int)));
HANDLE_ERROR(hipMalloc((void**)&outGlobe,DIM * DIM * sizeof (int)));
	
for (int i = 0; i < DIM*DIM; ++i)
{
	tmp[i] = i;	
}  

fun<<<DIM,DIM>>>(inGlobe,host_a);

for (int i = 0; i < DIM * DIM; ++i)	
		cout << host_a[i] << "  " ;
	

 int k;
 cin >> k;
return 0;
}


/*
After the host memory mapped to device, you can use hipHostGetDevicePointer
get the addr in device. then access the data transparent.
data transfer will be implicit. cuda core can optimize for it.
cuda core automatically overlap the data transfer and kernel exec.
*/



#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define DIM 16
__shared__ int share[256];

__global__ void fun(int * in, int * out)
{	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	//int  * dev_a;
	
	out[tid] = tid;
}

int main()
{
int * inGlobe;
int * outGlobe;
int tmp[DIM*DIM];
int * host_a;

HANDLE_ERROR(hipHostAlloc( (void**)&host_a,DIM * DIM * sizeof(int),
	hipHostMallocWriteCombined |
                              hipHostMallocPortable |
                              hipHostMallocMapped ));

for (int i = 0; i < DIM*DIM; ++i)
{
	host_a[i] = i;	
} 

HANDLE_ERROR(hipMalloc((void**)&inGlobe, DIM * DIM * sizeof (int)));
HANDLE_ERROR(hipMalloc((void**)&outGlobe,DIM * DIM * sizeof (int)));
	
for (int i = 0; i < DIM*DIM; ++i)
{
	tmp[i] = i;	
}

int * dev_out ;

HANDLE_ERROR(hipHostGetDevicePointer( &dev_out, host_a, 0 ));

fun<<<DIM,DIM>>>(inGlobe,dev_out);

for (int i = 0; i < DIM * DIM; ++i)	
		cout << host_a[i] << "  " ;
	

 int k;
 cin >> k;
return 0;
}

