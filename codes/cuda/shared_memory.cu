/*
example is to show how to use shared memory and every block has seperate
entity for shared memory. no impact on other block's shared memory.

and we can't use memcpy to set init value for shared memory since its addr
space not in global linear. 

but how to init it, not clear now.
*/

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

#define DIM 16
__shared__ int share[256];

__global__ void fun(int * in, int * out)
{
	
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < DIM)
	share[0] = tid;
    //from output we can see every block has its own entity for shared memory
	out[tid] = share[0];
}

int main()
{
int * inGlobe;
int * outGlobe;
int tmp[DIM*DIM];
int tmp2[DIM][DIM];

HANDLE_ERROR(hipMalloc((void**)&inGlobe, DIM * DIM * sizeof (int)));
HANDLE_ERROR(hipMalloc((void**)&outGlobe,DIM * DIM * sizeof (int)));
	
for (int i = 0; i < DIM*DIM; ++i)
{
	tmp[i] = i;		
	
}

HANDLE_ERROR(hipMemcpy( inGlobe, tmp, DIM * DIM * sizeof (int), hipMemcpyHostToDevice ));    

fun<<<DIM,DIM>>>(inGlobe,outGlobe);
	
HANDLE_ERROR(hipMemcpy( tmp2, outGlobe, DIM * DIM * sizeof (int), hipMemcpyDeviceToHost ));

for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
		cout << tmp2[i][j] << "  " ;
	cout << endl;
}
 int k;
 cin >> k;
return 0;
}

