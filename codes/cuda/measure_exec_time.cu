#include "stdafx.h"
#include <time.h>
#include <fstream>
#include "../common/book.h"
#define DIM 32 * 1024

int main( void ) {    
    // capture the start time
    hipEvent_t     start, stop;
    HANDLE_ERROR( hipEventCreate( &start ) );
    HANDLE_ERROR( hipEventCreate( &stop ) );
    HANDLE_ERROR( hipEventRecord( start, 0 ) );
	// get stop time, and display the timing results
    HANDLE_ERROR( hipEventRecord( stop, 0 ) );

    //hipEventSynchronize is important here, then CPU will block until 
    //cuda stream is complete.
    HANDLE_ERROR( hipEventSynchronize( stop ) );
    float   elapsedTime;
    HANDLE_ERROR( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );
    printf( "Time to generate:  %3.1f ms\n", elapsedTime );

    HANDLE_ERROR( hipEventDestroy( start ) );
    HANDLE_ERROR( hipEventDestroy( stop ) );
	return 0;
}

