#include "hip/hip_runtime.h"
/*
texture<DataType,Type,ReadMode>

Type:
hipTextureType1D
hipTextureType2D
hipTextureType3D
hipTextureType1DLayered
hipTextureType2DLayered

ReadMode:
hipReadModeElementType
hipReadModeNormalizedFloat

*/


#include <iostream>
#include "stdio.h"
using namespace std;
const int DIM = 16;
texture<float,2,hipReadModeElementType>  texIn;

__global__ void fun(float * in, float * out)
{
	int x = threadIdx.x;
	int y = threadIdx.y;	
	int tid = threadIdx.x + threadIdx.y * blockDim.x;
	out[tid] = tex2D(texIn,fx,fy) + 0.1;
	
}

int main()
{
float * inGlobe;
float * outGlobe;
float tmp1[DIM][DIM];
float tmp[DIM][DIM];
dim3    threads(DIM,DIM);

texIn.normalized = true;                      // access with normalized texture coordinates
    texIn.filterMode = hipFilterModeLinear;      // linear interpolation

hipMalloc((void**)&inGlobe,DIM * DIM );
hipMalloc((void**)&outGlobe,DIM * DIM * 4);
hipChannelFormatDesc desc = hipCreateChannelDesc<float>();
hipBindTexture2D( NULL, texIn,inGlobe,
                 desc, DIM, DIM,
                 DIM * sizeof (float) );
	
for (int i = 0; i < DIM; ++i)
	for (int j = 0; j < DIM; ++j)
		tmp1[i][j] = (i*DIM + j + 0.1) / 256;

hipMemcpy( inGlobe, tmp1, DIM * DIM * sizeof (float), hipMemcpyHostToDevice );    

fun<<<1,threads>>>(inGlobe,outGlobe);
	
hipMemcpy(tmp,outGlobe,  DIM * DIM * sizeof (float), hipMemcpyDeviceToHost );

for (int i = 0; i < DIM; ++i)
{
	for (int j = 0; j < DIM; ++j)
		printf("%f ", tmp[i][j]);
	cout << endl;
}
 int k;
 cin >> k;
return 0;
}

