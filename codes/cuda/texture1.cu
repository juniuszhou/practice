#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>  
    
#include <iostream>
#include "stdio.h"
#include <hip/hip_runtime.h>
    //#include <cudaGL.h>
    
    using namespace std;
    
    const int DIM = 16;
    texture<unsigned char,2,hipReadModeNormalizedFloat>  texIn;
    
    __global__ void fun(unsigned char * in, float * out)
    {
        int x = threadIdx.x;
        int y = threadIdx.y;    
        int tid = threadIdx.x + threadIdx.y * blockDim.x;   
        //out[tid] = tex2D(texIn,x,y) + in[tid];
        out[tid] = tex2D(texIn,x,y)  + in[tid] + x * 10000 + y * 1000000;
    }
    
    int main()
    {
    //memory in kernel
    unsigned char * inGlobe;
    float * outGlobe;
    
    //memory in host
    unsigned char tmp1[DIM][DIM];
    float tmp[DIM][DIM];
    
    dim3 threads(DIM,DIM);
    
    hipMalloc((void**)&inGlobe,DIM * DIM );
    hipMalloc((void**)&outGlobe,DIM  * DIM * sizeof (float));
    
    //hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
    hipChannelFormatDesc desc = hipCreateChannelDesc(16, 16, 0, 0, hipChannelFormatKindUnsigned);

    
    hipBindTexture2D( NULL, texIn,inGlobe,
                     desc, DIM, DIM,
                     DIM * DIM * sizeof (unsigned char) );
    //root cause is there is error, we need DIM * DIM * sizeof (char) 
        
    for (int i = 0; i < DIM; ++i)
        for (int j = 0;  j< DIM; ++j)
            tmp1[i][j] = i*16 + j;
    
    hipMemcpy( inGlobe, tmp1, DIM * DIM * sizeof(unsigned char), hipMemcpyHostToDevice );    
    
    fun<<<1,threads>>>(inGlobe,outGlobe);
    hipDeviceSynchronize();    
    hipMemcpy(tmp,outGlobe,  DIM *DIM * sizeof(float), hipMemcpyDeviceToHost );
    hipDeviceSynchronize();
    for (int i = 0; i < DIM; ++i)
    {
        for (int j = 0;  j< DIM; ++j)
        {   
            printf("%f ", tmp[i][j]);       
        }
        cout << endl;
    }
     int k;
     cin >> k;
    return 0;
    }
    











#include <hip/hip_runtime.h>      
#include <iostream>
#include "stdio.h"
#include <hip/hip_runtime.h>     
    using namespace std;
        
        const int DIM = 8;
        texture<unsigned char,2,hipReadModeNormalizedFloat>  texIn;
        
        __global__ void fun(unsigned char * in, float * out)
        {
            int x = threadIdx.x;
            int y = threadIdx.y;    
            int tid = threadIdx.x + threadIdx.y * blockDim.x;
            printf(" %d %d %d  %d %f \n  ",x,y,tid, in[tid],tex2D(texIn,x,y));
            //out[tid] = tex2D(texIn,x,y) + in[tid];
            //out[tid] = tex2D(texIn,x,y)  + in[tid] + x * 10000 + y * 1000000;
        }
        
        int main()
        {
        //memory in kernel
        unsigned char * inGlobe;
        float * outGlobe;
        
        //memory in host
        unsigned char tmp1[DIM][DIM];
        float tmp[DIM][DIM];
        
        dim3 threads(DIM,DIM);
        
        hipMalloc((void**)&inGlobe,DIM * DIM );
        hipMalloc((void**)&outGlobe,DIM  * DIM * sizeof (float));
        
        //hipChannelFormatDesc desc = hipCreateChannelDesc<unsigned char>();
        hipChannelFormatDesc desc = hipCreateChannelDesc(DIM, DIM, 0, 0, hipChannelFormatKindUnsigned);
    
        
        hipBindTexture2D( NULL, texIn,inGlobe,
                         desc, DIM, DIM,
                         sizeof (unsigned char) );
        //root cause is there is error, we need DIM * DIM * sizeof (char) 
            
        for (int i = 0; i < DIM; ++i)
            for (int j = 0;  j< DIM; ++j)
                tmp1[i][j] = i * DIM + j ;
        
        hipMemcpy( inGlobe, tmp1, DIM * DIM * sizeof(unsigned char), hipMemcpyHostToDevice );    
        
        fun<<<1,threads>>>(inGlobe,outGlobe);
        //hipDeviceSynchronize();    
        hipMemcpy(tmp,outGlobe,  DIM *DIM * sizeof(float), hipMemcpyDeviceToHost );
        hipDeviceSynchronize();
        /*
        for (int i = 0; i < DIM; ++i)
        {
            for (int j = 0;  j< DIM; ++j)
            {   
                printf("%f ", tmp[i][j]);       
            }
            cout << endl;
        }
        */
         int k;
         cin >> k;
         hipFree(inGlobe);
         hipFree(outGlobe);
        return 0;
        }
        
    
    

