/*
sample to show the matrix multiply. it is for square.
need a little bit modification for rectangle.
*/

#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
        cout <<  hipGetErrorString(err) << file << line << endl;
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

const int DIM = 4;
const int block_size = 4;

__global__ void multi(int * A, int * B, int * C)
{
   __shared__ int As[block_size][block_size];       
   __shared__ int Bs[block_size][block_size];

	// Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

	int len = DIM * block_size;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = len * by;

    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + len - 1;

    // Step size used to iterate through the sub-matrices of A
    int aStep  = block_size;

    // Index of the first sub-matrix of B processed by the block
    int bBegin = block_size * bx;

    // Step size used to iterate through the sub-matrices of B
    int bStep  = block_size * len;

    // Csub is used to store the element of the block sub-matrix
    // that is computed by the thread
    float Csub = 0;

    // Loop over all the sub-matrices of A and B
    // required to compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) 
	{        
        As[ty][tx] = A[a + len * ty + tx];
        Bs[ty][tx] = B[b + len * ty + tx];       
        __syncthreads();       

        for (int k = 0; k < block_size; ++k)
            Csub += As[ty][k] * Bs[k][tx];
		__syncthreads();
    }
   
    int c = len * (by * block_size + ty) + block_size * bx + tx;
    C[c] = Csub;	
}

int main()
{
int * inGlobe;
int * in2Globe;
int * outGlobe;

HANDLE_ERROR(hipMalloc((void**)&inGlobe,DIM * DIM * block_size * block_size * sizeof(int)));
HANDLE_ERROR(hipMalloc((void**)&in2Globe,DIM * DIM * block_size * block_size * sizeof(int)));
HANDLE_ERROR(hipMalloc((void**)&outGlobe,DIM * DIM * block_size * block_size * sizeof(int)));


int tmp[DIM*DIM* block_size * block_size];
for(int i=0;i<DIM*DIM;++i)
	for(int j=0;j<block_size * block_size;++j)
	tmp[i*DIM*DIM + j] = 2;

HANDLE_ERROR(hipMemcpy(inGlobe,tmp,DIM*DIM* block_size * block_size*sizeof(int),hipMemcpyHostToDevice));
for(int i=0;i<DIM*DIM;++i)
	for(int j=0;j<block_size * block_size;++j)
	tmp[i*DIM*DIM + j] = 5;

HANDLE_ERROR(hipMemcpy(in2Globe,tmp,DIM*DIM* block_size * block_size*sizeof(int),hipMemcpyHostToDevice));

for(int i=0;i<DIM*DIM;++i)
	for(int j=0;j<block_size * block_size;++j)
	tmp[i*DIM*DIM + j] = 0;

HANDLE_ERROR(hipMemcpy(outGlobe,tmp,DIM*DIM* block_size * block_size*sizeof(int),hipMemcpyHostToDevice));

dim3 bdim(DIM,DIM);
dim3 tdim(block_size,block_size);

multi<<<bdim,tdim>>>(inGlobe,in2Globe,outGlobe);
HANDLE_ERROR(hipMemcpy(tmp,outGlobe,DIM*DIM* block_size * block_size*sizeof(int),hipMemcpyDeviceToHost));
hipDeviceSynchronize();

for(int i=0;i<DIM*DIM;++i)
{
	for(int j=0;j<block_size * block_size;++j)	
	   cout << " " << tmp[i*DIM*DIM + j];
	cout << endl;
}


 int k;
 cin >> k;
return 0;
}

